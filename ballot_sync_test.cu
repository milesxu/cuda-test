#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void ballot_sync_test() {
  auto test = 0;
  if (threadIdx.x % 7 == 0) {
    test = 1;
  }
  auto result = __ballot_sync(0xffffffff, test);
  printf("%d\n", result);
}

int main(int argc, char *argv[]) {
  ballot_sync_test<<<1, 32>>>();
  hipDeviceSynchronize();
  return 0;
}